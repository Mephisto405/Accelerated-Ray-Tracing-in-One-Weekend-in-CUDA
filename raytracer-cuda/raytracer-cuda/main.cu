#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <time.h>
#include "vec3.h"
#include "ray.h"

using namespace std;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__device__ vec3 sky(const ray& r) {
	vec3 u = unit_vector(r.direction());
	float w = 0.5f * (u.y() + 1.0f);
	return (1.0f - w) * vec3(1.0, 1.0, 1.0) + w * vec3(0.5, 0.7, 1.0);
}

__global__ void render(vec3 *fb, int max_x, int max_y,
						vec3 origin, vec3 lower_left_corner, vec3 horizontal, vec3 vertical) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;

	/// normalized uv coordinate
	float u = float(i) / float(max_x);
	float v = float(j) / float(max_y);

	ray r(origin, lower_left_corner + u*horizontal + v*vertical);
	fb[pixel_index] = sky(r);
}

int main() {
	/*
	Initiliaztion
	*/
	int nx = 1200;
	int ny = 600;
	int tx = 8;
	int ty = 8;
	vec3 lower_left_corner(-2.0, -1.0, -1.0);
	vec3 horizontal(4.0, 0.0, 0.0);
	vec3 vertical(0.0, 2.0, 0.0);
	vec3 origin(0.0, 0.0, 0.0);

	std::cerr << "Rendering a " << nx << "x" << ny << " image ";
	std::cerr << "in " << tx << "x" << ty << " blocks.\n";

	int num_pixels = nx*ny;
	size_t fb_size = num_pixels*sizeof(vec3);

	// File open
	ofstream file;
	file.open("image.ppm");

	// allocate FB
	vec3 *fb;
	checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

	clock_t start, stop;
	start = clock();

	/*
	Setting thread blocks and Run
	*/
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render<<<blocks, threads>>>(fb, nx, ny, origin, 
								lower_left_corner, horizontal, vertical);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	stop = clock();
	double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
	std::cerr << "took " << timer_seconds << " seconds.\n";

	/*
	Output FB as Image
	*/
	file << "P3\n" << nx << " " << ny << "\n255\n";
	for (int j = ny - 1; j >= 0; j--) {
		for (int i = 0; i < nx; i++) {
			size_t pixel_index = j * nx + i;
			vec3 col = fb[pixel_index];
			int ir = int(255.99f*col[0]);
			int ig = int(255.99f*col[1]);
			int ib = int(255.99f*col[2]);
			file << ir << " " << ig << " " << ib << "\n";
		}
	}

	file.close();
	checkCudaErrors(hipFree(fb));
}